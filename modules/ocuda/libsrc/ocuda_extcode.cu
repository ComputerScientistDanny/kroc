/*
 *	ocuda_extcode.cu -- occam-pi/CUDA interface glue code
 *	Copyright (C) 2013 Fred Barnes <frmb@kent.ac.uk>
 *
 *	This program is free software; you can redistribute it and/or modify
 *	it under the terms of the GNU General Public License as published by
 *	the Free Software Foundation; either version 2 of the License, or
 *	(at your option) any later version.
 *
 *	This program is distributed in the hope that it will be useful,
 *	but WITHOUT ANY WARRANTY; without even the implied warranty of
 *	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *	GNU General Public License for more details.
 *
 *	You should have received a copy of the GNU General Public License
 *	along with this program; if not, write to the Free Software
 *	Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA  02110-1301 USA
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

#include <dmem_if.h>

#include "ocuda_ctypes.h"

#define MAX_GPU_INSTANCES	(8)
#undef OCUDA_DEBUG

/* per-thread context */
static ocuda_devinfo_t devinfo[MAX_GPU_INSTANCES];

int __get_last_cuda_error (const char *msg, const char *file, const int line) /*{{{*/
{
	hipError_t err = hipGetLastError ();

	if (err != hipSuccess) {
		fprintf (stderr, "%s(%d): CUDA error: %s: (%d): %s\n", file, line, msg, (int)err, hipGetErrorString (err));
		return 1;
	}
	return 0;
}
/*}}}*/


static inline void real_cuda_init (int *d_count, int *sp) /*{{{*/
{
	int ndevices, i, dcnt;

	*sp = get_last_cuda_error ("initialise");
	if (*sp) {
		return;
	}

	hipGetDeviceCount (&ndevices);
	if (get_last_cuda_error ("hipGetDeviceCount()")) {
		return;
	}

	if (*d_count == 0) {
		*d_count = ndevices;
	}

	for (i=0,dcnt=0; dcnt<(*d_count); dcnt++) {
		hipDeviceProp_t prop;
		int j;

		hipGetDeviceProperties (&prop, dcnt);
		if (get_last_cuda_error ("hipGetDeviceProperties()")) {
			continue;					/* just skip this one */
		}

		hipSetDevice (dcnt);					/* create stream(s) in device context */
		devinfo[i].dnum = dcnt;
		// hipStreamCreate (&(devinfo[i].stream0));
		// hipStreamCreate (&(devinfo[i].stream1));
		// devinfo[i].stream0 = 0;
		// devinfo[i].stream1 = 0;
		// if (get_last_cuda_error ("cudaCreateStream()")) {
		// 	return;
		// }

		devinfo[i].cversion = ((prop.major << 16) | (prop.minor & 0xffff));
		if (strlen (prop.name) > 64) {
			memcpy (devinfo[i].dname, prop.name, 64);
			devinfo[i].dname_len = 64;
		} else {
			devinfo[i].dname_len = strlen (prop.name);
			memcpy (devinfo[i].dname, prop.name, devinfo[i].dname_len);
		}
		devinfo[i].cmode = prop.computeMode;
		devinfo[i].global_mem = (prop.totalGlobalMem / 1024);
		devinfo[i].shared_mem_per_block = (prop.sharedMemPerBlock / 1024);
		devinfo[i].regs_per_block = prop.regsPerBlock;
		devinfo[i].warp_size = prop.warpSize;
		devinfo[i].mem_pitch = prop.memPitch;
		devinfo[i].max_threads_per_block = prop.maxThreadsPerBlock;
		for (j=0; j<3; j++) {
			devinfo[i].max_threads_dim[j] = prop.maxThreadsDim[j];
			devinfo[i].max_grid_size[j] = prop.maxGridSize[j];
		}
		devinfo[i].clock_rate = prop.clockRate;
		devinfo[i].total_const_mem = (prop.totalConstMem / 1024);
		devinfo[i].mproc_count = prop.multiProcessorCount;
		devinfo[i].max_threads_per_mproc = prop.maxThreadsPerMultiProcessor;
		devinfo[i].integrated = prop.integrated;
		devinfo[i].can_map_host_memory = prop.canMapHostMemory;
		devinfo[i].concur_kernels = prop.concurrentKernels;
		devinfo[i].async_engine_count = prop.asyncEngineCount;
		devinfo[i].pci_bus_id = prop.pciBusID;
		devinfo[i].pci_device_id = prop.pciDeviceID;
		devinfo[i].pci_domain_id = prop.pciDomainID;
		devinfo[i].tcc_driver = prop.tccDriver;
		devinfo[i].unified_addressing = prop.unifiedAddressing;
		devinfo[i].mem_clock_rate = prop.memoryClockRate;
		devinfo[i].mem_bus_width = prop.memoryBusWidth;
		devinfo[i].l2_cache_size = (prop.l2CacheSize / 1024);

#if defined(OCUDA_DEBUG)
		fprintf (stderr, "CUDA device %d: \"%s\" (%d.%d) init\n", devinfo[i].dnum, prop.name, (devinfo[i].cversion >> 16) & 0xffff, (devinfo[i].cversion & 0xffff));
#endif
		i++;
	}
	*d_count = i;

	return;
}
/*}}}*/
static inline void real_cuda_devinfo (int dnum, ocuda_devinfo_t *result, int *res) /*{{{*/
{
	if ((dnum < 0) || (dnum >= MAX_GPU_INSTANCES)) {
		memset (result, 0, sizeof (ocuda_devinfo_t));
		*res = 1;
		return;
	}
	memcpy (result, &(devinfo[dnum]), sizeof (ocuda_devinfo_t));
	*res = 0;
}
/*}}}*/

extern "C" {
	/* PROC C.cuda.init (INT d.count, RESULT INT res) */
	__host__ void _cuda_init (int *ws) { real_cuda_init ((int *)(ws[0]), (int *)(ws[1])); }

	/* PROC C.cuda.devinfo (VAL INT dnum, RESULT OCUDA.DEVINFO data, RESULT INT res) */
	__host__ void _cuda_devinfo (int *ws) { real_cuda_devinfo ((int)(ws[0]), (ocuda_devinfo_t *)(ws[1]), (int *)(ws[2])); }
}


