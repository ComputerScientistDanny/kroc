/*
 *	ocuda_extcode.cu -- occam-pi/CUDA interface glue code
 *	Copyright (C) 2013 Fred Barnes <frmb@kent.ac.uk>
 *
 *	This program is free software; you can redistribute it and/or modify
 *	it under the terms of the GNU General Public License as published by
 *	the Free Software Foundation; either version 2 of the License, or
 *	(at your option) any later version.
 *
 *	This program is distributed in the hope that it will be useful,
 *	but WITHOUT ANY WARRANTY; without even the implied warranty of
 *	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *	GNU General Public License for more details.
 *
 *	You should have received a copy of the GNU General Public License
 *	along with this program; if not, write to the Free Software
 *	Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA  02110-1301 USA
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "ocuda_ctypes.h"

#define MAX_GPU_INSTANCES	(8)

/* per-thread context */
static ocuda_devinfo_t devinfo[MAX_GPU_INSTANCES];

int __get_last_cuda_error (const char *msg, const char *file, const int line) /*{{{*/
{
	hipError_t err = hipGetLastError ();

	if (err != hipSuccess) {
		fprintf (stderr, "%s(%d): CUDA error: %s: (%d): %s\n", file, line, msg, (int)err, hipGetErrorString (err));
		return 1;
	}
	return 0;
}
/*}}}*/

#define get_last_cuda_error(MSG) __get_last_cuda_error (MSG, __FILE__, __LINE__)

static inline void real_cuda_init (int *d_count, int *sp) /*{{{*/
{
	int ndevices, i;

	*sp = get_last_cuda_error ("initialise");
	if (*sp) {
		return;
	}

	hipGetDeviceCount (&ndevices);
	if (get_last_cuda_error ("hipGetDeviceCount()")) {
		return;
	}

	if (*d_count == 0) {
		*d_count = ndevices;
	}

	for (i=0; i<(*d_count); i++) {
		hipDeviceProp_t prop;

		hipGetDeviceProperties (&prop, i);
		if (get_last_cuda_error ("hipGetDeviceProperties()")) {
			return;
		}

		hipSetDevice (i);					/* create stream(s) in device context */
		devinfo[i].dnum = i;
		// hipStreamCreate (&(devinfo[i].stream0));
		// hipStreamCreate (&(devinfo[i].stream1));
		// devinfo[i].stream0 = 0;
		// devinfo[i].stream1 = 0;
		// if (get_last_cuda_error ("cudaCreateStream()")) {
		// 	return;
		// }

		devinfo[i].cversion = ((prop.major << 16) | (prop.minor & 0xffff));
		if (strlen (prop.name) > 64) {
			memcpy (devinfo[i].dname, prop.name, 64);
			devinfo[i].dname_len = 64;
		} else {
			devinfo[i].dname_len = strlen (prop.name);
			memcpy (devinfo[i].dname, prop.name, devinfo[i].dname_len);
		}
		fprintf (stderr, "CUDA device %d: \"%s\" (%d.%d) init\n", devinfo[i].dnum, prop.name, (devinfo[i].cversion >> 16) & 0xffff, (devinfo[i].cversion & 0xffff));
	}

	return;
}
/*}}}*/
static inline void real_cuda_devinfo (int dnum, ocuda_devinfo_t *result, int *res) /*{{{*/
{
	if ((dnum < 0) || (dnum >= MAX_GPU_INSTANCES)) {
		memset (result, 0, sizeof (ocuda_devinfo_t));
		*res = 1;
		return;
	}
	memcpy (result, &(devinfo[dnum]), sizeof (ocuda_devinfo_t));
	*res = 0;
}
/*}}}*/

extern "C" {
	/* PROC C.cuda.init (INT d.count, RESULT INT res) */
	__host__ void _cuda_init (int *ws) { real_cuda_init ((int *)(ws[0]), (int *)(ws[1])); }
	/* PROC C.cuda.devinfo (VAL INT dnum, RESULT OCUDA.DEVINFO data, RESULT INT res) */
	__host__ void _cuda_devinfo (int *ws) { real_cuda_devinfo ((int)(ws[0]), (ocuda_devinfo_t *)(ws[1]), (int *)(ws[2])); }
}


