#include "hip/hip_runtime.h"
/*
 *	mandelcuda.cu -- CUDA mandelbrot code for use with occam-pi
 *	Copyright (C) 2013 Fred Barnes <frmb@kent.ac.uk>
 *	Derived from earlier occam-pi Mandelbrot code by Jim Moores and David Wood.
 *
 *	This program is free software; you can redistribute it and/or modify
 *	it under the terms of the GNU General Public License as published by
 *	the Free Software Foundation; either version 2 of the License, or
 *	(at your option) any later version.
 *
 *	This program is distributed in the hope that it will be useful,
 *	but WITHOUT ANY WARRANTY; without even the implied warranty of
 *	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *	GNU General Public License for more details.
 *
 *	You should have received a copy of the GNU General Public License
 *	along with this program; if not, write to the Free Software
 *	Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA  02110-1301 USA
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

#include <dmem_if.h>
#include <ocuda_ctypes.h>

#define MANDEL_ABSSQLIM 32.0

/*{{{  mandelcuda_ainfo_t: type to define various device-specific settings that can be passed to/from occam*/
typedef struct TAG_mandelcuda_ainfo {
	ocuda_devinfo_t *device;		/* ocuda device structure */
	int *dev_fb;				/* device allocated framebuffer memory */
	int width;				/* width of fractal in pixels */
	int height;				/* height of fractal in pixels */
	int fbsize;				/* dev_fb size in bytes */
	double *dev_parms;			/* device allocated parameters memory */
	int parmssize;				/* dev_parms size in bytes */
	int *dev_iparms;			/* device allocated integer parameters memory */
	int iparmssize;				/* dev_iparms size in bytes */
	int *dev_colour;			/* device allocated colouring-in array */
	int coloursize;				/* dev_colour size in bytes */
} mandelcuda_ainfo_t;


/*}}}*/
/*{{{  complex64_t: 64-bit complex number*/
typedef struct TAG_complex64 {
	double real;
	double imag;
} complex64_t;

/*}}}*/


__device__ inline double complex64abssq (const complex64_t c) /*{{{*/
{
	return (c.real * c.real) + (c.imag * c.imag);
}
/*}}}*/
__device__ inline complex64_t complex64add (const complex64_t x, const complex64_t y) /*{{{*/
{
	complex64_t tmp;

	tmp.real = x.real + y.real;
	tmp.imag = x.imag + y.imag;

	return tmp;
}
/*}}}*/
__device__ inline complex64_t complex64mul (const complex64_t x, const complex64_t y) /*{{{*/
{
	complex64_t tmp;

	tmp.real = (x.real * y.real) - (x.imag * y.imag);
	tmp.imag = (x.imag * y.real) + (x.real * y.imag);
	return tmp;
}
/*}}}*/
__device__ inline int mandelbrot_calc (const complex64_t c) /*{{{*/
{
	int count = 1;
	complex64_t iter = c;

	while ((complex64abssq (iter) < MANDEL_ABSSQLIM) && (count < 256)) {
		iter = complex64add (complex64mul (iter, iter), c);
		count++;
	}
	return count;
}
/*}}}*/

/*{{{  __global__ void mandelbrot_kernel (const double *dparms, int *framebuffer, const int *iparms, const int *colouring)*/
/*
 *	This implements the GPU kernel, extracts various settings from parameters, calculates a single pixel and plants in framebuffer.
 */
__global__ void mandelbrot_kernel (const double *dparms, int *framebuffer, const int *iparms, const int *colouring)
{
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int width = iparms[0];
	int height = iparms[1];
	double y = (double)(j - (height / 2));
	double x = (double)(i - (width / 2));
	complex64_t c;
	int val, pixel;

	y = (y / dparms[2]) - dparms[1];
	x = (x / (dparms[2] * 2.0)) - dparms[0];
	c.real = x;
	c.imag = y;

	val = mandelbrot_calc (c);
	if (val >= 256) {
		val = 255;
	}
	pixel = colouring[(val + iparms[2]) & 0xff];
	/*
	pixel = 0xf0000000 >> (val * 2);

	if (pixel & 0xffffff) {
		// rotate low-order 24 bits by iparms[2]
		pixel = (pixel >> iparms[2]) | (pixel << (24 - iparms[2]));
	}
	*/

	framebuffer[(j * width) + i] = pixel;
}

/*}}}*/

static inline void real_cuda_allocmem (ocuda_devinfo_t *device, void **ainfo, const int *diparms) /*{{{*/
{
	mandelcuda_ainfo_t *minfo = (mandelcuda_ainfo_t *)dmem_alloc (sizeof (mandelcuda_ainfo_t));

	minfo->device = device;
	minfo->width = diparms[0];
	minfo->height = diparms[1];
	minfo->fbsize = minfo->width * minfo->height * sizeof (int);
	minfo->parmssize = 3 * sizeof (double);
	minfo->iparmssize = 3 * sizeof (int);
	minfo->coloursize = 240 * sizeof (int);

	hipSetDevice (minfo->device->dnum);

	hipMalloc ((void **)&minfo->dev_fb, minfo->fbsize);
	hipMalloc ((void **)&minfo->dev_parms, minfo->parmssize);
	hipMalloc ((void **)&minfo->dev_iparms, minfo->iparmssize);
	hipMalloc ((void **)&minfo->dev_colour, minfo->coloursize);

	*ainfo = (void *)minfo;
}

/*}}}*/
static inline void real_cuda_freemem (void **ainfo) /*{{{*/
{
	mandelcuda_ainfo_t *minfo = (mandelcuda_ainfo_t *)(*ainfo);

	if (!minfo) {
		return;
	}

	hipSetDevice (minfo->device->dnum);

	hipFree (minfo->dev_colour);
	hipFree (minfo->dev_iparms);
	hipFree (minfo->dev_parms);
	hipFree (minfo->dev_fb);

	dmem_release (minfo);
	*ainfo = NULL;
}

/*}}}*/
static inline void real_cuda_updatecolour (void **ainfo, const int *colouring) /*{{{*/
{
	mandelcuda_ainfo_t *minfo = (mandelcuda_ainfo_t *)(*ainfo);

	hipSetDevice (minfo->device->dnum);
	hipMemcpy (minfo->dev_colour, colouring, minfo->coloursize, hipMemcpyHostToDevice);
	get_last_cuda_error ("hipMemcpy(HostToDevice)");
}
/*}}}*/


static inline void real_cuda_mandelbrot (void **ainfo, const double *dparms, int *framebuffer, const int *diparms) /*{{{*/
{
	mandelcuda_ainfo_t *minfo = (mandelcuda_ainfo_t *)(*ainfo);
	int ydim = 16;
	int xdim = (minfo->device->max_threads_per_block / ydim);

	hipSetDevice (minfo->device->dnum);
	hipMemcpy (minfo->dev_parms, dparms, minfo->parmssize, hipMemcpyHostToDevice);
	hipMemcpy (minfo->dev_iparms, diparms, minfo->iparmssize, hipMemcpyHostToDevice);
	get_last_cuda_error ("hipMemcpy(HostToDevice)");

	dim3 dim_block (xdim, ydim);
	dim3 dim_grid (minfo->width / dim_block.x, minfo->height / dim_block.y);

	mandelbrot_kernel <<< dim_grid, dim_block >>> (minfo->dev_parms, minfo->dev_fb, minfo->dev_iparms, minfo->dev_colour);

	hipMemcpy (framebuffer, minfo->dev_fb, minfo->fbsize, hipMemcpyDeviceToHost);
	get_last_cuda_error ("hipMemcpy(DeviceToHost)");
}

/*}}}*/


extern "C" {
	/* PROC C.cuda.allocmem (OCUDA.DEVINFO device, MANDELCUDA.AINFO ainfo, VAL []INT iparams) */
	__host__ void _cuda_allocmem (int *ws) { real_cuda_allocmem ((ocuda_devinfo_t *)(ws[0]), (void **)(ws[1]), (int *)(ws[2])); }

	/* PROC C.cuda.freemem (MANDELCUDA.AINFO ainfo) */
	__host__ void _cuda_freemem (int *ws) { real_cuda_freemem ((void **)(ws[0])); }

	/* PROC C.cuda.updatecolour (MANDELCUDA.AINFO ainfo, VAL []INT colouring) */
	__host__ void _cuda_updatecolour (int *ws) { real_cuda_updatecolour ((void **)(ws[0]), (int *)(ws[1])); }

	/* PROC C.cuda.mandelbrot (MANDELCUDA.AINFO ainfo, VAL []REAL64 settings, [][]INT fb, VAL []INT iparams) */
	__host__ void _cuda_mandelbrot (int *ws) {
		real_cuda_mandelbrot ((void **)(ws[0]), (double *)(ws[1]), (int *)(ws[3]), (int *)(ws[6])); }
}

